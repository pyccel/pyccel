#include "cuda_ndarrays.h"

void    device_memory(void** devPtr, size_t size)
{
    hipMalloc(devPtr, size);
}

void    managed_memory(void** devPtr, size_t size)
{
    hipMallocManaged(devPtr, size);
}

void    host_memory(void** devPtr, size_t size)
{
    hipHostMalloc(devPtr, size);
}
t_cuda_ndarray   cuda_array_create(enum e_memory_locations location, int32_t nd, int64_t *shape,
        enum cu_types type, bool is_view)
{
    t_cuda_ndarray arr;
    void (*fun_ptr_arr[])(void**, size_t) = {managed_memory, host_memory, device_memory};

    arr.nd = nd;
    arr.type = type;
    switch (type)
    {
        case cu_int8:
            arr.type_size = sizeof(int8_t);
            break;
        case cu_int16:
            arr.type_size = sizeof(int16_t);
            break;
        case cu_int32:
            arr.type_size = sizeof(int32_t);
            break;
        case cu_int64:
            arr.type_size = sizeof(int64_t);
            break;
        case cu_float:
            arr.type_size = sizeof(float);
            break;
        case cu_double:
            arr.type_size = sizeof(double);
            break;
        case cu_bool:
            arr.type_size = sizeof(bool);
            break;
    }
    arr.is_view = is_view;
    arr.length = 1;
    arr.shape = (int64_t *)malloc(arr.nd * sizeof(int64_t));
    for (int32_t i = 0; i < arr.nd; i++)
    {
        arr.length *= shape[i];
        arr.shape[i] = shape[i];
    }
    arr.buffer_size = arr.length * arr.type_size;

    if (!is_view)
        (*fun_ptr_arr[location])(&(arr.raw_data), arr.buffer_size);
    return (arr);
}

int32_t cuda_free_host(t_cuda_ndarray arr)
{
    if (arr.shape == NULL)
        return (0);
    hipHostFree(arr.raw_data);
    arr.raw_data = NULL;
    hipFree(arr.shape);
    arr.shape = NULL;
    return (1);
}

__host__ __device__
int32_t cuda_free(t_cuda_ndarray arr)
{
    if (arr.shape == NULL)
        return (0);
    hipFree(arr.raw_data);
    arr.raw_data = NULL;
    hipFree(arr.shape);
    arr.shape = NULL;
    return (0);
}

__host__ __device__
int32_t cuda_free_pointer(t_cuda_ndarray arr)
{
    if (arr.is_view == false || arr.shape == NULL)
        return (0);
    hipFree(arr.shape);
    arr.shape = NULL;
    return (0);
}
